#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void test_cu(unsigned int* mem_buffer, uint len) {
    uint temp = 0;
    __asm__ __volatile__("s_getreg_b32 %0 hwreg(HW_REG_HW_ID)\ns_nop 2\n" : "=s"(temp));
    if (blockIdx.x < len && threadIdx.x == 0) {
        mem_buffer[blockIdx.x] = temp;
    }
}

static uint32_t se_from_hwid(uint32_t hwid) {
    return (hwid >> 13) & 0x7;
}

static uint32_t cu_from_hwid(uint32_t hwid) {
    return (hwid >> 8) & 0xf;
}

static uint32_t simd_from_hwid(uint32_t hwid) {
    return (hwid >> 4) & 0x3;
}

int main() {
    const uint buffer_size = 120;
    unsigned int* d_mem_buffer;

    // Allocate memory on the GPU
    hipMalloc(&d_mem_buffer, buffer_size * sizeof(unsigned int));

    // Launch the kernel
    dim3 grid(buffer_size);
    dim3 block(64);
    test_cu<<<grid, block>>>(d_mem_buffer, buffer_size);

    // Retrieve the results from the GPU
    unsigned int h_mem_buffer[buffer_size];
    hipMemcpy(h_mem_buffer, d_mem_buffer, buffer_size * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Print decoded information
    for (uint i = 0; i < buffer_size; ++i) {
        uint32_t hwid = h_mem_buffer[i];
        std::cout << "Blockidx= " << i << ", Shader Engine# (0-7)=" << se_from_hwid(hwid)
                  << ", local_CU_idx_within_SE# (not consecutive; few CUs within SE maybe disabled in any order)=" << cu_from_hwid(hwid) 
                  << ", SIMD=" << simd_from_hwid(hwid) << std::endl;
    }

    // Clean up
    hipFree(d_mem_buffer);
    return 0;
}
